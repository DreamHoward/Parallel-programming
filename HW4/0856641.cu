/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void update (void);
void printfinal (void);

int nsteps,                 	 /* number of time steps */
    tpoints, 	     		         /* total points along string */
    rcode;                  	 /* generic return code */

float  values[MAXPOINTS+2], 	 /* values at time t */
       old_val[MAXPOINTS+2], 	 /* values at time (t-dt) */
       new_val[MAXPOINTS+2]; 	 /* values at time (t+dt) */


float  *device_value;

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n",
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);
}


__global__ void parallel(float *device_value, int tpoints, int nsteps)
{
    // init_line() 
    int i, k;
    int blockSize = 1024;
    float value, x, fac, tmp, old_val;
    
    fac = 2.0 * PI;
    k = 1 + blockIdx.x * blockSize + threadIdx.x;
    tmp = tpoints - 1;
    x = (k - 1) / tmp;
    value = sin (fac * x);
    old_val = value;

    // do_math() 
    float dtime, c, dx, tau, sqtau;
    
    dtime = 0.3;
    c = 1.0;
    dx = 1.0;
    tau = (c * dtime / dx);
    sqtau = tau * tau;

    // update() 
    float new_val;
    
    if(k <= tpoints) {
      for (i = 1; i<= nsteps; i++) {
        if ((k == 1) || (k  == tpoints))
          new_val = 0.0;
        else
          new_val = (2.0 * value) - old_val + (sqtau * -2.0 * value);
        old_val = value;
        value = new_val;
      }
      device_value[k] = value;
    }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i % 10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
    sscanf(argv[1], "%d", &tpoints);
  	sscanf(argv[2], "%d", &nsteps);
    check_param();
    
    int size = (tpoints + 1) * sizeof(float);
    int blockSize = 1024;
    int block_num = tpoints / blockSize + !(tpoints % blockSize == 0);
    
    //cuda malloc
    hipMalloc((void**) &device_value, size);

    printf("Initializing points on the line...\n");
  	printf("Updating all points for all time steps...\n");
    //parallel
    parallel<<<block_num, blockSize>>>(device_value, tpoints, nsteps);

    hipMemcpy(values, device_value, size, hipMemcpyDeviceToHost);
    hipFree(device_value);

    printf("Printing final results...\n");
  	printfinal();
  	printf("\nDone.\n\n");

	return 0;
}